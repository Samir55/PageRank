#include "hip/hip_runtime.h"
/*
 * Kernel.cpp
 *
 *  Created on: Feb 27, 2018
 *      Author: Ahmed Samir
 */

#include "Kernel.hpp"

namespace PageRank {

// A Kernel for multiplying square matrices.
    __global__ void page_rank_iteration (Page* d_pages, double* d_page_probs, int* d_edges_list, int pages_count, double d_dangling_sum, double alpha) {
        int idx = blockIdx.y * blockDim.y + threadIdx.y;

        double c_element = 0.0;
        int i_start = d_pages[idx].start_idx;
        int i_end = d_pages[idx].start_idx;

        for (int i = i_start; i < i_end; i++) {
            // Get the index of current node linking to this node
            int from = d_edges_list[i];

            c_element += d_page_probs[from] * 1 / d_pages[from].out_links_count; 
        }

        c_element = (1 - alpha) * 1.0 / pages_count + alpha * (c_element + d_dangling_sum / pages_count);

        __syncthreads();

        d_page_probs[idx] = c_element;
    }

    void Kernel::run_kernel(int dangling_nodes_count) {
        // Calculate the grid and block sizes.
        int grid_size = int(ceil(1.0 * pages_count / MAX_BLOCK_SIZE));
        int block_size = int(ceil(1.0 * pages_count / grid_size));

        if (block_size < 1024) {
            dim3 dimGrid(1, grid_size);
            dim3 dimBlock(1, block_size);

            for (int i = 0; i < MAX_ITERATIONS; ++i) {
                page_rank_iteration << < dimGrid, dimBlock >> > (d_pages, d_pages_probs, d_edges_list, pages_count, dangling_nodes_count, ALPHA);
            }
        } else {
            cout << "Error exceeded the maximum value for threads in a block 1024" << endl;
        }
    }

    void Kernel::allocate_data(Page* h_pages, double* h_pages_probs, int* h_edges_list) {
        // Allocate memory at the gpu device
        hipMalloc ((void **) &d_pages, sizeof(Page) * pages_count);
        hipMalloc ((void **) &d_pages_probs, sizeof(double) * pages_count);
        hipMalloc ((void **) &d_edges_list, sizeof(int) * edges_count);

        // Copy data from host (cpu) to the gpu
        hipMemcpy(d_pages, h_pages, sizeof(Page) * pages_count, hipMemcpyDeviceToHost);
        hipMemcpy(d_pages_probs, h_pages_probs, sizeof(double) * pages_count, hipMemcpyDeviceToHost);
        hipMemcpy(d_edges_list, h_edges_list, sizeof(int) * edges_count, hipMemcpyDeviceToHost);
    }

    double* Kernel::get_result() {
        double* pages_probs = new double[pages_count];

        hipMemcpy(pages_probs, d_pages_probs, sizeof(double) * pages_count, hipMemcpyDeviceToHost);
               hipError_t e = hipGetLastError();
                if (e != hipSuccess) {
                    printf("MemCpy (R): CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
                    exit(0);
                }

        return pages_probs;
    }

} /* namespace PageRank */
