#include "hip/hip_runtime.h"
/*
 * Kernel.cpp
 *
 *  Created on: Feb 27, 2018
 *      Author: Ahmed Samir
 */

#include "Kernel.h"

namespace PageRank {

// A Kernel for multiplying square matrices.
    __global__ void page_rank_iteration(Matrix d_a, Matrix d_b, Matrix d_c, double* d_sum, int n, double alpha) {
        double c_element = 0.0;

        int idx = blockIdx.y * blockDim.y + threadIdx.y;
        if (idx < n) {
            for (int i = 0; i < n; i++) {
                c_element += (d_a[idx * n + i] * d_b[i]);
            }
            d_c[idx] = (alpha * c_element) + (1.0 - alpha) * 1.0/n;
        }
    }

    __global__ void update_i_vector(Matrix d_b, Matrix d_c) {
        int idx = blockIdx.y * blockDim.y + threadIdx.y;

        // Normalize the vector.
        // copy the resulted vector from c to b for re multiplying and final result of course
        // after the final iteration will be stored in d_c
        d_b[idx] = d_c[idx];
    }

    void Kernel::run_kernel() {
        // Calculate the grid and block sizes.
        int grid_size = int(ceil(1.0 * n / MAX_BLOCK_SIZE));
        int block_size = int(ceil(1.0 * n / grid_size));

        if (block_size < 1024) {
            dim3 dimGrid(1, grid_size);
            dim3 dimBlock(1, block_size);

            for (int i = 0; i < MAX_ITERATIONS; ++i)
            {
                page_rank_iteration<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, d_sum, n, ALPHA);
                update_i_vector<<<dimGrid, dimBlock>>>(d_b, d_c);
            }
        } else {
            cout << "Error exceeded the maximum value for threads in a block 1024" << endl;
        }
    }

    void Kernel::allocate_matrices(Matrix h_a, Matrix h_b) {
        long long matirx_bytes = sizeof(double) * n * n;
        long long vector_bytes = sizeof(double) * n;

        double* h_sum = new double;
        *h_sum = 1.0;


        // Allocate memory at the device for matrices a, b, and the result c
        hipMalloc((void **) &d_a, matirx_bytes);
        hipMalloc((void **) &d_b, vector_bytes);
        hipMalloc((void **) &d_c, vector_bytes);
        hipMalloc((void **) &d_sum, sizeof(double));

        // Copy matrices a & b to the device
        hipMemcpy(d_a, h_a, matirx_bytes, hipMemcpyHostToDevice);
        hipError_t e=hipGetLastError();
        if(e!=hipSuccess) {
            printf("MemCpy (A): CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
            exit(0);
        }

        hipMemcpy(d_b, h_b, vector_bytes, hipMemcpyHostToDevice);
        e =hipGetLastError();
        if(e!=hipSuccess) {
            printf("MemCpy (B): CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
            exit(0);
        }

        hipMemcpy(d_sum, h_sum, sizeof(double), hipMemcpyHostToDevice);
        e =hipGetLastError();
        if(e!=hipSuccess) {
            printf("MemCpy (B): CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
            exit(0);
        }
        delete h_sum;

    }

    Matrix Kernel::get_result() {

        Matrix h_c = new double[n];

        int vector_bytes = sizeof(double) * n;

        hipMemcpy(h_c, d_c, vector_bytes, hipMemcpyDeviceToHost);
        hipError_t e=hipGetLastError();
        if(e!=hipSuccess) {
            printf("MemCpy (R): CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
            exit(0);
        }

        return h_c;
    }

} /* namespace PageRank */
