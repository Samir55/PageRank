#include "hip/hip_runtime.h"
/*
 * Kernel.cpp
 *
 *  Created on: Feb 27, 2018
 *      Author: Ahmed Samir
 */

#include "Kernel.hpp"

namespace PageRank {

__global__ void initialize_dangling_sum(float *d_dangling_probs_sum) {
	d_dangling_probs_sum[0] = 0.0;
}

__global__ void initialize_pages_ranks_sum(float *d_page_ranks_sum) {
	d_page_ranks_sum[0] = 0.0;
}

__global__ void calculate_dangling_sum_and_normalize (Page *d_pages, double *d_page_probs, int pages_count, float *d_page_ranks_sum, float *d_dangling_probs_sum) {
	// Calculate page index from thread address
	int idx = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < pages_count) {
		d_page_probs[idx] /= d_page_ranks_sum[0];
		if (d_pages[idx].dangling_node) {
			atomicAdd(d_dangling_probs_sum, float(d_page_probs[idx]));
		}
	}
}

__global__ void run_page_rank_iteration(Page *d_pages,
		double *d_page_probs,
		int *d_edges_list,
		int pages_count,
		float *d_pages_ranks_sum,
		float *d_dangling_probs_sum,
		double alpha) {

	int idx = blockIdx.y * blockDim.y + threadIdx.y;
	double new_rank = 0.0;

	if (idx < pages_count) {
		double c_element = 0.0;
		int i_start = d_pages[idx].start_idx;
		int i_end = d_pages[idx].end_idx;

		for (int i = i_start; i < i_end; i++) {
			int from = d_edges_list[i];
			c_element += d_page_probs[from] / ( 1.0 * d_pages[from].out_links_count);
		}

		new_rank = ((1 - alpha) * 1.0 / pages_count)+
				(alpha * c_element) +
				(alpha * 1.0 / pages_count * d_dangling_probs_sum[0]);
	}

	__syncthreads();

	if (idx < pages_count) {
		d_page_probs[idx] = new_rank;
		atomicAdd(d_pages_ranks_sum, new_rank);
	}
}

void Kernel::run_kernel() {
	// Calculate the grid and block sizes.
	int grid_size = int(ceil(1.0 * pages_count / MAX_BLOCK_SIZE));
	int block_size = MAX_BLOCK_SIZE;

	if (block_size < 1024) {
		dim3 dimGrid(1, grid_size);
		dim3 dimBlock(1, block_size);

		for (int i = 0; i < MAX_ITERATIONS; ++i) {
			if (i > 0) {
				initialize_dangling_sum<<<1, 1>>>(d_dangling_probs_sum);
				calculate_dangling_sum_and_normalize << <dimGrid, dimBlock>> >(d_pages, d_pages_probs, pages_count, d_pages_ranks_sum, d_dangling_probs_sum);
			}

			initialize_pages_ranks_sum<<<1, 1>>>(d_pages_ranks_sum);
			run_page_rank_iteration << < dimGrid, dimBlock >> > (d_pages, d_pages_probs, d_edges_list, pages_count, d_pages_ranks_sum, d_dangling_probs_sum, ALPHA);
		}
	} else {
		cout << "Error exceeded the maximum value for threads in a block 1024" << endl;
	}
}

void Kernel::allocate_data(Page *h_pages, double *h_pages_probs, int *h_edges_list) {
	float one = 1.0, zero = 0.0;

	// Allocate memory at the gpu device
	hipMalloc((void **) &d_pages, sizeof(Page) * pages_count);
	hipMalloc((void **) &d_pages_probs, sizeof(double) * pages_count);
	hipMalloc((void **) &d_edges_list, sizeof(int) * edges_count);
	hipMalloc((void **) &d_pages_ranks_sum, sizeof(float));
	hipMalloc((void **) &d_dangling_probs_sum, sizeof(float));

	// Copy data from host (cpu) to the gpu
	hipMemcpy(d_pages, h_pages, sizeof(Page) * pages_count, hipMemcpyHostToDevice);
	hipMemcpy(d_pages_probs, h_pages_probs, sizeof(double) * pages_count, hipMemcpyHostToDevice);
	hipMemcpy(d_edges_list, h_edges_list, sizeof(int) * edges_count, hipMemcpyHostToDevice);
	hipMemcpy(d_pages_ranks_sum, &one, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dangling_probs_sum, &zero, sizeof(float), hipMemcpyHostToDevice);
}

double *Kernel::get_result() {
	double *pages_probs = new double[pages_count];

	hipMemcpy(pages_probs, d_pages_probs, sizeof(double) * pages_count, hipMemcpyDeviceToHost);
	hipError_t e = hipGetLastError();
	if (e != hipSuccess) {
		printf("MemCpy (R): CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
		exit(0);
	}

	return pages_probs;
}

} /* namespace PageRank */
